#define N 1024
#define M 2
#define S 1

#include "../src/utils/cuda/utils_cublas.cu"
#include "../src/config.cpp"

int main (int argc, char const *argv[])
{
	double* a;
	double* b;
	
	std::vector <cuda::utils::vector <double> > a_devs (S);
	std::vector <cuda::utils::vector <double> > b_devs (S);
	
	hipEvent_t start, stop;
	float elapsed_time;
	
	std::vector<hipStream_t> streams (S);
	
	HANDLE_ERROR (hipEventCreate (&start));
	HANDLE_ERROR (hipEventCreate (&stop));
	HANDLE_ERROR (hipEventRecord (start, 0));
	
	for (int i = 0; i < S; ++i) {
		hipStreamCreate (&streams [i]);
		a_devs [i].resize (N);
		b_devs [i].resize (N);
	}
		
	HANDLE_ERROR (hipHostAlloc (&a, S * M * N * sizeof (double), hipHostMallocDefault));
	HANDLE_ERROR (hipHostAlloc (&b, S * M * N * sizeof (double), hipHostMallocDefault));
	
	for (int i = 0; i < S * M * N; ++i) {
		a [i] = (double) i;
		b [i] = (double) (i * i);
	}
	
	for (int i = 0; i < S * M * N; ++i) {
		INFO ("In: " << a [i] << " " << b [i]);
	}
	
	for (int i = 0; i < M * S * N; i += S * N) {
		for (int j = 0; j < S; ++j) {
			HANDLE_ERROR (hipMemcpyAsync (a_devs [j].pointer (), a + j * N + i, N * sizeof (double), hipMemcpyHostToDevice, streams [j]));
			HANDLE_ERROR (hipMemcpyAsync (b_devs [j].pointer (), b + j * N + i, N * sizeof (double), hipMemcpyHostToDevice, streams [j]));
		}

		for (int j = 0; j < S; ++j) {
			cublasSetKernelStream (streams [j]);
			for (int k = 0; k < 20000; ++k) {
				cuda::utils::add_scaled (N, 1.0, a_devs [j].pointer (), b_devs [j].pointer ());
			}
		}

		for (int j = 0; j < S; ++j) {
			HANDLE_ERROR (hipMemcpyAsync (b + i, b_devs [j].pointer (), N * sizeof (double), hipMemcpyDeviceToHost, streams [j]));
		}
	}
	
	HANDLE_ERROR (hipEventRecord (stop, 0));
	
	HANDLE_ERROR (hipEventSynchronize (stop));
	HANDLE_ERROR (hipEventElapsedTime (&elapsed_time, start, stop));
	
	for (int i = 0; i < S; ++i) {
		HANDLE_ERROR (hipStreamSynchronize (streams [i]));
		HANDLE_ERROR (hipStreamDestroy (streams [i]));
	}
	
	for (int i = 0; i < S * M * N; ++i) {
		INFO ("Out: " << b [i]);
	}
	
	INFO ("Elapsed: " << elapsed_time << " ms.");
	
	HANDLE_ERROR (hipHostFree (a));
	HANDLE_ERROR (hipHostFree (b));
	
	return 0;
}