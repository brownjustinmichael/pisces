/*!**********************************************************************
 * \file utils_cublas.cu
 * /Users/justinbrown/Dropbox/pisces
 * 
 * Created by Justin Brown on 2013-08-07.
 * Copyright 2013 Justin Brown. All rights reserved.
 ************************************************************************/

#include "../../config.hpp"
#include "utils_cublas.hpp"
#include "utils_cuda.hpp"
#include <vector>
#include <cassert>
#include <hipblas.h>
#include <stdio.h>

namespace utils
{
	namespace cuda
	{
		namespace cublas
		{
			struct config
			{
			public:
				config () {
					CUBLAS_HANDLE_ERROR (cublasInit ());
				}
	
				virtual ~config () {
					CUBLAS_HANDLE_ERROR (cublasShutdown ());
				}
			};
	
			void copy (int n, float* x, float* y, int incx, int incy) {
				hipblasScopy (n, x, incx, y, incy);
			}

			void copy (int n, double* x, double* y, int incx, int incy) {
				hipblasDcopy (n, x, incx, y, incy);
			}

			void scale (int n, float a, float* x, int incx) {
				hipblasSscal (n, a, x, incx);
			}
	
			void scale (int n, double a, double* x, int incx) {
				hipblasDscal (n, a, x, incx);
			}

			double dot (int n, float* x, float* y, int incx, int incy) {
				return hipblasSdot (n, x, incx, y, incy);
			}

			double dot (int n, double* x, double* y, int incx, int incy) {
				return hipblasDdot (n, x, incx, y, incy);
			}

			void add_scaled (int n, float a, float* x, float* y, int incx, int incy) {
				hipblasSaxpy (n, a, x, incx, y, incy);
			}

			void add_scaled (int n, double a, double* x, double* y, int incx, int incy) {
				hipblasDaxpy (n, a, x, incx, y, incy);
			}

			void matrix_vector_multiply (int m, int n, float alpha, float* a, float* x, float beta, float* y, int lda, int incx, int incy) {
				char charN = 'N';
	
				assert (x != y);
	
				if (lda == -1) {
					lda = m;
				}
		
				hipblasSgemv (charN, m, n, alpha, a, lda, x, incx, beta, y, incy);
			}
	
			void matrix_vector_multiply (int m, int n, double alpha, double* a, double* x, double beta, double* y, int lda, int incx, int incy) {
				char charN = 'N';
	
				assert (x != y);
	
				if (lda == -1) {
					lda = m;
				}
		
				hipblasDgemv (charN, m, n, alpha, a, lda, x, incx, beta, y, incy);
			}
	
			config config_instance;
		} /* cublas */
	} /* cuda */
} /* utils */

