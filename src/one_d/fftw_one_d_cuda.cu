#include "hip/hip_runtime.h"
/*!**********************************************************************
 * \file fftw_one_d_cuda.cu
 * /Users/justinbrown/Dropbox/spectral_element
 * 
 * Created by Justin Brown on 2013-08-16.
 * Copyright 2013 Justin Brown. All rights reserved.
 ************************************************************************/

#include <math.h>
#include <hipfft/hipfft.h>
#include "fftw_one_d_cuda.hpp"
#include "../utils/utils_cublas.cuh"

#define HANDLE_ERROR(status) \
{hipError_t result = status; \
switch (result) { \
	case hipErrorOutOfMemory: FATAL ("Memory Allocation Error."); throw 0; \
	case hipErrorInvalidValue: FATAL ("Invalid value passed."); throw 0; \
	default: if (status != hipSuccess) {FATAL ("Other problem."); throw 0;}}}

#define HANDLE_CUFFT(status) \
{hipfftResult result = status; \
switch (result) { \
	case HIPFFT_INVALID_PLAN: FATAL ("Invalid plan for cufft."); throw 0; \
	case HIPFFT_INVALID_VALUE: FATAL ("Invalid value for cufft."); throw 0; \
	case HIPFFT_INTERNAL_ERROR: FATAL ("Internal driver error for cufft."); throw 0; \
	case HIPFFT_EXEC_FAILED: FATAL ("Failed to execute transform on cufft."); throw 0; \
	default: if (status != HIPFFT_SUCCESS) {FATAL ("Cufft Other problem."); throw 0;}}}


__global__ void real_to_complex (int n, double* in, hipfftDoubleComplex* out) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n) {
		out [tid].x = in [tid];
		out [tid].y = 0.0;
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void complex_to_real (int n, hipfftDoubleComplex* in, double* out) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n) {
		out [tid] = in [tid].x;
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void symmetrize (int n, double* data) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < n - 1 && tid != 0) {
		data [2 * n - 2 - tid] = data [tid];
		tid += blockDim.x * gridDim.x;
	}
}

namespace one_d
{
	namespace cuda
	{
		fftw_cosine::fftw_cosine (bases::element <double>* i_element_ptr, int i_n, int i_name_in, int i_name_out) : 
		bases::explicit_plan <double> (i_element_ptr, i_n, i_name_in, i_name_out) {
			TRACE ("Instantiating...");
			HANDLE_ERROR (hipMalloc ((void **) &data_real, 2 * n * sizeof (hipfftDoubleReal)));
			HANDLE_ERROR (hipMalloc ((void **) &data_complex, n * sizeof (hipfftDoubleComplex)));
			cu_plan = new hipfftHandle;
			HANDLE_CUFFT (hipfftPlan1d(cu_plan, 2 * n - 2, HIPFFT_D2Z, 1));
			scalar = sqrt (1.0 / 2.0 / ((double) n - 2.0));
			TRACE ("Instantiated.");
		}
		
		fftw_cosine::~fftw_cosine () {
			hipfftDestroy (*cu_plan);
			// delete cu_plan;
			hipFree (data_real);
			hipFree (data_complex);
		}
		
		void fftw_cosine::execute () {
			bases::explicit_plan <double>::execute ();
			std::vector <hipfftDoubleComplex> temp (n);
			
			HANDLE_ERROR (hipMemcpy (data_real, data_in, n * sizeof (double), hipMemcpyHostToDevice));
			
			for (int i = 0; i < n; ++i) {
				DEBUG ("Transforming: " << data_in [i]);
			}
			
			symmetrize <<<1, std::min (n, 512)>>> (n, data_real);
			
			HANDLE_ERROR (hipDeviceSynchronize ());
			
			HANDLE_CUFFT (hipfftPlan1d(cu_plan, 2 * n - 2, HIPFFT_D2Z, 1));
			/* Use the CUFFT plan to transform the signal in place. */
			HANDLE_CUFFT (hipfftExecD2Z(*cu_plan, data_real, data_complex));

			hipMemcpy (&temp [0], data_complex, n * sizeof (hipfftDoubleComplex), hipMemcpyDeviceToHost);
			
			for (int i = 0; i < n; ++i) {
				DEBUG ("REAL: " << temp [i].x);
				DEBUG ("IMAG: " << temp [i].y);
			}
			
			complex_to_real <<<1, std::min (n, 512)>>> (n, data_complex, data_real);
					
			hipMemcpy (data_out, data_real, n * sizeof (double), hipMemcpyDeviceToHost);
			
			for (int i = 0; i < n; ++i) {
				DEBUG ("Transformed: " << data_out [i]);
			}
			
			for (int i = 0; i < n; ++i) {
				data_out [i] *= scalar;
			}
		}
	} /* cuda */
} /* one_d */
