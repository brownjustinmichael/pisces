/*!**********************************************************************
 * \file solver_one_d_cuda.cu
 * /Users/justinbrown/Dropbox/spectral_element/src
 * 
 * Created by Justin Brown on 2013-08-21.
 * Copyright 2013 Justin Brown. All rights reserved.
 ************************************************************************/

#include "solver_one_d_cuda.hpp"
#include "fftw_one_d_cuda.hpp"
#include "../../utils/cuda/solver_utils_cuda.hpp"

namespace one_d
{
	namespace cuda
	{
		template <class datatype>
		solver <datatype>::solver (bases::element <datatype>* i_element_ptr, int i_n, int i_excess_0, int i_excess_n, datatype& i_timestep, datatype& i_alpha_0, datatype& i_alpha_n, datatype *i_default_matrix, datatype *i_matrix, int i_name_in, int i_name_rhs, int i_name_out = null, int i_flags = 0x00) :
		one_d::solver <datatype> (i_element_ptr, i_n, i_excess_0, i_excess_n, i_timestep, i_alpha_0, i_alpha_n, i_default_matrix, i_matrix, i_name_in, i_name_rhs, i_name_out, i_flags) {
			HANDLE_ERROR (hipMalloc ((void**) &factorized_matrix_dev, n * n * sizeof (datatype)));
			HANDLE_ERROR (hipMalloc ((void**) &ipiv_dev, n * sizeof (int)));
		}
		
		template <class datatype>
		solver <datatype>::~solver () {
			HANDLE_ERROR (hipFree (factorized_matrix_dev));
			HANDLE_ERROR (hipFree (ipiv_dev));
		}
		
		template <class datatype>
		void solver <datatype>::_factorize () {
			one_d::solver <datatype>::_factorize ();
			
			HANDLE_ERROR (hipMemcpy (factorized_matrix_dev, &factorized_matrix [0], n * n * sizeof (datatype), hipMemcpyHostToDevice));
			HANDLE_ERROR (hipMemcpy (ipiv_dev, &ipiv [0], n * sizeof (int), hipMemcpyHostToDevice));
		}
		
		template <class datatype>
		void solver <datatype>::execute () {
			utils::cuda::copy (n, data_in, data_out);
		
			data_out [0] += alpha_0 * timestep * rhs [0];
			data_out [n - 1] += alpha_n * timestep * rhs [n - 1];
			utils::add_scaled (n - 2, timestep, rhs + 1, data_out + 1);
			
			utils::cuda::matrix_solve (n, factorized_matrix_dev, ipiv_dev, data_out);
		}
		
		template class solver <double>;
	} /* cuda */
} /* one_d */